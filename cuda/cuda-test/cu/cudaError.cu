#include <iostream>

#include "hipError_t.cuh"

void CudaFetchError() {
  hipError_t cudaerr = hipDeviceSynchronize();
  if (cudaerr != hipSuccess)
    std::cout << "kernel launch failed with error "
              << hipGetErrorString(cudaerr) << std::endl;
}