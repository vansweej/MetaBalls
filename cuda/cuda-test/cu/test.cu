#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <functional>
#include <iostream>
#include <memory>

#include "cuda_error.cuh"
#include "test.cuh"
#include "cuda_properties_iterator.h"


int maxGridSize[3];
int maxThreadPerBlock;

__global__ void cuda_hello() { printf("Hello World from GPU!\n"); }

void GetCudaProperties() {
  CudaProperties cudaProperties;
  for (auto prop : cudaProperties) {
    std::cout << prop.name << std::endl;
  }
}

void add() {
  const unsigned int N = 101;
  int a[N], b[N], result[N], result_2[N];
  int *dev_a, *dev_b, *dev_result;

  CUDA_ERROR(hipMalloc((void **)&dev_a, N * sizeof(int)));
  CUDA_ERROR(hipMalloc((void **)&dev_b, N * sizeof(int)));
  CUDA_ERROR(hipMalloc((void **)&dev_result, N * sizeof(int)));

  std::for_each(std::begin(a), std::end(a), [](int &v) {
    static int i = 1;
    v = i;
    i++;
  });
  std::for_each(std::begin(b), std::end(b), [](int &v) {
    static int i = 1;
    v = i * i;
    i++;
  });

  std::transform(std::begin(a), std::end(a), b, result_2, std::minus<>{});

  CUDA_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
  CUDA_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));

  int threads =
      std::max(1, std::min(maxThreadPerBlock, int(std::ceil(sqrt(N)))));
  int blocks = std::max(1, int(std::ceil(N / (float)threads)));

  add<<<blocks, threads>>>(dev_a, dev_b, dev_result, N);

  CUDA_ERROR(
      hipMemcpy(result, dev_result, N * sizeof(int), hipMemcpyDeviceToHost));

  std::for_each(std::begin(result), std::end(result),
                [&result, &result_2](const int &r) {
                  static int c = 0;
                  assert(result[c] == result_2[c]);
                  std::cout << r << std::endl;
                  c++;
                });

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_result);
}

__global__ void add(int *dev_a, int *dev_b, int *dev_result, int N) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  // printf("N = %d | tid = %d\n", N, tid);
  while (tid < N) {
    dev_result[tid] = dev_a[tid] - dev_b[tid];
    tid += blockDim.x * gridDim.x;
  }
}